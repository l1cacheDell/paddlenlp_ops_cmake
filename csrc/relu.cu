#include "hip/hip_runtime.h"
#include "paddle/extension.h"

template <typename data_t>
__global__ void relu_cuda_forward_kernel(const data_t* x,
                                         data_t* y,
                                         int64_t num) {
  int64_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (int64_t i = gid; i < num; i += blockDim.x * gridDim.x) {
    y[i] = max(x[i], static_cast<data_t>(0.));
  }
}

std::vector<paddle::Tensor> relu_cuda_forward(const paddle::Tensor& x) {

  auto out = paddle::empty_like(x);

  int64_t numel = x.numel();
  int64_t block = 512;
  int64_t grid = (numel + block - 1) / block;
  PD_DISPATCH_FLOATING_TYPES(
      x.type(), "relu_cuda_forward_kernel", ([&] {
        relu_cuda_forward_kernel<data_t><<<grid, block, 0, x.stream()>>>(
            x.data<data_t>(), 
            out.data<data_t>(), 
            numel);
      }));

  return {out};
}

// 维度推导
std::vector<std::vector<int64_t>> ReluInferShape(std::vector<int64_t> x_shape) {
  return {x_shape};
}

// 类型推导
std::vector<paddle::DataType> ReluInferDtype(paddle::DataType x_dtype) {
  return {x_dtype};
}

PD_BUILD_OP(custom_relu)
    .Inputs({"X"})
    .Outputs({"Out"})
    .SetKernelFn(PD_KERNEL(relu_cuda_forward))
    .SetInferShapeFn(PD_INFER_SHAPE(ReluInferShape))
    .SetInferDtypeFn(PD_INFER_DTYPE(ReluInferDtype));